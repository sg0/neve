#include "hip/hip_runtime.h"
#include "types.hpp"
#include "cuda_wrapper.hpp"
#include <hip/hip_cooperative_groups.h> 

namespace cg = cooperative_groups;

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long* addr_as_ull = (unsigned long long*)address;
    unsigned long long  old = *addr_as_ull;
    unsigned long long  assumed;
    do 
    {
        assumed = old;
        old = atomicCAS(addr_as_ull, assumed,__double_as_longlong(__longlong_as_double(assumed)+val));
    } while(assumed != old);

    return __longlong_as_double(old);
}
#endif

template<const int WarpSize, const int BlockSize>
__global__
void reorder_weights_by_keys_kernel
(
    GraphWeight* __restrict__ edgeWeights,
    GraphElem*   __restrict__ indexOrders,
    GraphElem*   __restrict__ indices,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem ranges[(BlockSize/WarpSize)*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid  = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += (BlockSize/WarpSize)*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();

        GraphElem start = t_ranges[0]-e_base; 
        GraphElem end   = t_ranges[1]-e_base;
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
        {
            GraphElem pos = indexOrders[i];
            #ifdef USE_32BIT_GRAPH
            indexOrders[i] = __float_as_int(edgeWeights[pos+start]);
            #else
            indexOrders[i] = __double_as_longlong(edgeWeights[pos+start]);
            #endif
        }
        warp.sync();
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
            #ifdef USE_32BIT_GRAPH
            edgeWeights[i] = __uint_as_float(indexOrders[i]);
            #else
            edgeWeights[i] = __longlong_as_double(indexOrders[i]);
            #endif
        warp.sync();
    } 
}


void reorder_weights_by_keys_cuda
( 
    GraphWeight* edgeWeights, 
    GraphElem* indexOrders, 
    GraphElem* indices, 
    const GraphElem& v0, 
    const GraphElem& v1,
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    GraphElem nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE); 
    reorder_weights_by_keys_kernel<WARPSIZE,BLOCKDIM01><<<nblocks, BLOCKDIM01,0,stream>>>(edgeWeights, indexOrders, indices, v0, e0, nv); 
}

template<const int WarpSize, const int BlockSize>
__global__
void reorder_edges_by_keys_kernel
(
    GraphElem* __restrict__ edges,
    GraphElem* __restrict__ indexOrders,
    GraphElem* __restrict__ indices,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem ranges[(BlockSize/WarpSize)*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid  = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += (BlockSize/WarpSize)*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();

        GraphElem start = t_ranges[0]-e_base; 
        GraphElem end = t_ranges[1]-e_base;
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
        {
            GraphElem pos = indexOrders[i];
            indexOrders[i] = edges[pos+start];
        }
        warp.sync();
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
            edges[i] = indexOrders[i];
        warp.sync();
    } 
}

void reorder_edges_by_keys_cuda
(
    GraphElem* edges, 
    GraphElem* indexOrders, 
    GraphElem* indices, 
    const GraphElem& v0, 
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    long long nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);
    reorder_edges_by_keys_kernel<WARPSIZE,BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(edges, indexOrders, indices, v0, e0, nv);
}

template<const int WarpSize, const int BlockSize>
__global__
void fill_edges_community_ids_kernel
(
    GraphElem2* __restrict__ commIdKeys,
    GraphElem*  __restrict__ edges,
    GraphElem*  __restrict__ indices,
    GraphElem*  __restrict__ commIds,
    const GraphElem v_base, 
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem ranges[BlockSize/WarpSize*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;

    for(GraphElem u = u0; u < nv; u += (GraphElem)BlockSize/WarpSize*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();
 
        GraphElem start = t_ranges[0] + warp_tid-e_base;               
        GraphElem end = t_ranges[1]-e_base;
        for(GraphElem i = start; i < end; i += WarpSize)
        {
            GraphElem commId = commIds[edges[i]];    
            #ifdef USE_32BIT_GRAPH
            commIdKeys[i] = make_int2(u, commId);
            #else
            commIdKeys[i] = make_longlong2(u, commId);
            #endif
        }
        warp.sync();
    } 
}

void fill_edges_community_ids_cuda
(
    GraphElem2* commIdKeys, 
    GraphElem* edges,
    GraphElem* indices,
    GraphElem* commIds,
    const GraphElem& v0,
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    long long nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);    
    fill_edges_community_ids_kernel<WARPSIZE, BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>
    (commIdKeys, edges, indices, commIds, v0, e0, nv);
}

template<const int WarpSize, const int BlockSize>
__global__
void fill_index_orders_kernel
(
    GraphElem* __restrict__ indexOrders,
    GraphElem* __restrict__ indices,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv 
)
{
    __shared__ GraphElem ranges[BlockSize/WarpSize*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;

    for(GraphElem u = u0; u < nv; u += (GraphElem)BlockSize/WarpSize*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();
 
        GraphElem start = t_ranges[0];               
        GraphElem end = t_ranges[1];
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
             indexOrders[i-e_base] = i-start;
        warp.sync();
    } 
}

void fill_index_orders_cuda
(
    GraphElem* indexOrders,
    GraphElem* indices,
    const GraphElem& v0,
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    int nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);    
    fill_index_orders_kernel<WARPSIZE, BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>
    (indexOrders, indices, v0, e0, nv);
}

#if 0
template<const int WarpSize, const int blocksize>
__global__
void sum_vertex_weights_kernel
(GraphWeight* __restrict__ vertex_degree, GraphWeight* __restrict__ weights,
GraphElem* __restrict__ edge_indices, const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem range[2];
    volatile  __shared__ GraphWeight data[blocksize];
    GraphElem step = gridDim.x;
    for(GraphElem i = blockIdx.x; i < nv; i += step)
    {
        GraphWeight w = 0.;
        if(threadIdx.x < 2)
            range[threadIdx.x] = edge_indices[i+threadIdx.x+v_base];
        __syncthreads();

        GraphElem start = range[0]-e_base;
        GraphElem end = range[1]-e_base;

        //GraphElem start = edge_indices[i];
        //GraphElem end = edge_indices[i+1];
        for(GraphElem e = start+threadIdx.x; e < end; e += blocksize)
            w += weights[e];
        data[threadIdx.x] = w;
        __syncthreads();
        for (unsigned int s=blocksize/2; s>=32; s>>=1)
        {
            if (threadIdx.x < s)
                data[threadIdx.x] += data[threadIdx.x + s];
            __syncthreads();
        }
        w = data[threadIdx.x%32];
        //__syncthreads();
        for (int offset = 16; offset > 0; offset /= 2)
            w += __shfl_down_sync(0xffffffff, w, offset);

        if(threadIdx.x == 0)
            vertex_degree[i+v_base] = w;
        __syncthreads();
    }
}
#endif
//#if 0
template<const int WarpSize, const int BlockSize>
__global__
void sum_vertex_weights_kernel
(
    GraphWeight* __restrict__ vertex_weights, 
    GraphWeight* __restrict__ weights,
    GraphElem*   __restrict__ indices,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{   
    __shared__ GraphElem ranges[BlockSize/WarpSize*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WarpSize> warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[block_tid/WarpSize*2];

    GraphElem step = gridDim.x*BlockSize/WarpSize;
    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += step)
    {   
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();
       
        GraphElem start = t_ranges[0]-e_base;
        GraphElem   end = t_ranges[1]-e_base;
        GraphWeight w = 0.; 
        for(GraphElem e = start+warp_tid; e < end; e += WarpSize)
            w += weights[e];
        warp.sync();

        for(int i = warp.size()/2; i > 0; i/=2)
            w += warp.shfl_down(w, i);
 
        if(warp_tid == 0) 
            vertex_weights[u+v_base] = w;
        warp.sync();
    }
}
//#endif

#if 0
template<const int WarpSize, const int BlockSize>
__global__
void sum_vertex_weights_kernel
(
    GraphWeight* __restrict__ vertex_weights, 
    GraphWeight* __restrict__ weights,
    GraphElem*   __restrict__ indices,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{   
    __shared__ GraphElem ranges[BlockSize];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WarpSize> warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*WarpSize];

    GraphElem nu = (nv + gridDim.x*(BlockSize/WarpSize)-1) / (gridDim.x*(BlockSize/WarpSize));
    GraphElem u0 = (BlockSize/WarpSize*blockIdx.x+block_tid/WarpSize)*nu;
    nu = ((u0+nu > nv) ? nv-u0 : nu);

    GraphElem start, end;
    start = indices[u0+v_base]-e_base;

    for(GraphElem u = 0; u < nu; ++u)
    {
        if(u%WarpSize == 0)
        {
            warp.sync();
            if(u+warp_tid < nu)
                t_ranges[warp_tid] = indices[u0+u+warp_tid+v_base+1];
            warp.sync();
        }
        end = t_ranges[u%WarpSize]-e_base;

        GraphWeight w = 0.; 
        for(GraphElem e = start+warp_tid; e < end; e += WarpSize)
            w += weights[e];
        warp.sync();

        for(int i = warp.size()/2; i > 0; i/=2)
            w += warp.shfl_down(w, i);
 
        if(warp_tid == 0) 
            vertex_weights[u+u0+v_base] = w;
        start = end;
    }
}
#endif

void sum_vertex_weights_cuda
(
    GraphWeight* vertex_weights,
    GraphWeight* weights,
    GraphElem*   indices,
    const GraphElem& v0,
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
     GraphElem nv = v1-v0;
     GraphElem nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE); 
     sum_vertex_weights_kernel<WARPSIZE,BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(vertex_weights, weights, indices, v0, e0, nv);
}

template<const int BlockSize>
__global__
void compute_community_weighted_orders_kernel
(
    GraphWeight* __restrict__ commWeights,
    GraphElem*   __restrict__ commIds,
    GraphWeight* __restrict__ vertexWeights,
    const GraphElem nv
)
{
    GraphElem u0 = threadIdx.x+BlockSize*blockIdx.x;
    for(GraphElem i = u0; i < nv; i += BlockSize*gridDim.x)
    {
        GraphElem comm_id = commIds[i];
        GraphWeight w = vertexWeights[i];
        atomicAdd(commWeights+comm_id, w);
    }
}

void compute_community_weights_cuda
(
    GraphWeight* commWeights,
    GraphElem* commIds, 
    GraphWeight* vertexWeights,
    const GraphElem& nv,
    hipStream_t stream = 0
)
{
    GraphElem nblocks = (nv > MAX_GRIDDIM) ? MAX_GRIDDIM : nv;
    compute_community_weighted_orders_kernel<BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(commWeights, commIds, vertexWeights, nv);
}

template<const int BlockSize>
__global__ 
void singleton_partition_kernel
(
    GraphElem*   __restrict__ commIds,
    GraphWeight* __restrict__ commWeights,
    GraphWeight* __restrict__ vertexWeights,
    const GraphElem nv
)
{
    GraphElem u0 = threadIdx.x + BlockSize*blockIdx.x;
    for(GraphElem i = u0; i < nv; i += BlockSize*gridDim.x)
    {
        commIds[i] = i;
        commWeights[i] = vertexWeights[i];
    }    
}

void singleton_partition_cuda
(
    GraphElem* commIds, 
    GraphWeight* commWeights, 
    GraphWeight* vertexWeights, 
    const GraphElem& nv, 
    hipStream_t stream = 0
)
{
    GraphElem nblocks = (nv > MAX_GRIDDIM) ? MAX_GRIDDIM : nv;
    singleton_partition_kernel<BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(commIds, commWeights, vertexWeights, nv);
}

template<const int BlockSize>
__global__
void scan_edges_kernel
(
    GraphElem* __restrict__ edges,
    Edge*      __restrict__ edgeList,
    const GraphElem ne
)
{
    GraphElem u0 = threadIdx.x + BlockSize*blockIdx.x;
    for(GraphElem i = u0; i < ne; i += BlockSize*gridDim.x)
    {
        Edge e = edgeList[i];
        edges[i] = e.tail_;
    }
}

void scan_edges_cuda
(
    GraphElem* edges, 
    Edge* edgeList, 
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem ne = e1-e0;
    long long nblocks = (ne > MAX_GRIDDIM) ? MAX_GRIDDIM : ne;
    scan_edges_kernel<BLOCKDIM01><<<nblocks,BLOCKDIM01,0,stream>>>(edges, edgeList, ne);
}

template<const int BlockSize>
__global__
void scan_edge_weights_kernel
(
    GraphWeight* __restrict__ edgeWeights,
    Edge*        __restrict__ edgeList,
    const GraphElem ne
)
{
    GraphElem u0 = threadIdx.x + BlockSize*blockIdx.x;
    for(GraphElem i = u0; i < ne; i += BlockSize*gridDim.x)
    {
        Edge e = edgeList[i];
        edgeWeights[i] = e.weight_;
    }
}

void scan_edge_weights_cuda
(
    GraphWeight* edgeWeights, 
    Edge* edgeList, 
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem ne = e1-e0;
    long long nblocks = (ne > MAX_GRIDDIM) ? MAX_GRIDDIM : ne;
    scan_edge_weights_kernel<BLOCKDIM01><<<nblocks,BLOCKDIM01,0,stream>>>(edgeWeights, edgeList, ne);
}

template<const int WarpSize, const int BlockSize>
__global__
void max_vertex_weights_kernel
(
    GraphWeight* __restrict__ maxVertexWeights,
    GraphWeight* __restrict__ edgeWeights,
    GraphElem*   __restrict__ edge_indices,
    const GraphElem v_base,   
    const GraphElem e_base,
    const GraphElem nv 
)
{
    __shared__ GraphElem ranges[BlockSize/WarpSize*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WarpSize> warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[block_tid/WarpSize*2];

    GraphElem step = gridDim.x*BlockSize/WarpSize;
    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += step)
    {   
        if(warp_tid < 2)
            t_ranges[warp_tid] = edge_indices[u+warp_tid+v_base];
        warp.sync();
       
        GraphElem start = t_ranges[0]-e_base;
        GraphElem   end = t_ranges[1]-e_base;
        volatile GraphWeight w = 0.; 
        for(GraphElem e = start+warp_tid; e < end; e += WarpSize)
        {
            GraphWeight tmp = edgeWeights[e];
            w = (tmp > w) ? tmp : w;
        }
        warp.sync();

        for(int i = warp.size()/2; i > 0; i/=2)
        {
            GraphWeight tmp = warp.shfl_down(w, i);
            w = (tmp > w) ? tmp : w;
        }
        if(warp_tid == 0) 
            maxVertexWeights[u+v_base] = w;
        warp.sync();
    }
}

void max_vertex_weights_cuda
(
    GraphWeight* maxVertexWeights,
    GraphWeight* edgeWeights,
    GraphElem* indices,
    const GraphElem& v0,
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    long long nblocks = (nv > MAX_GRIDDIM) ? MAX_GRIDDIM : nv;
    max_vertex_weights_kernel<WARPSIZE, BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(maxVertexWeights, edgeWeights, indices, v0, e0, nv);
}

template<const int BlockSize>
__global__
void max_order_reduce_kernel
(
    GraphElem* __restrict__ orders,
    GraphElem* __restrict__ indices, 
    GraphElem nv
)
{
    __shared__ GraphElem max_shared[BlockSize];

    max_shared[threadIdx.x] = 0;

    GraphElem u0 = threadIdx.x + BlockSize * blockIdx.x; 

    for(GraphElem u = u0; u < nv; u += BlockSize*gridDim.x)
    {    
        GraphElem order = indices[u+1]-indices[u];
        if(max_shared[threadIdx.x] < order) 
            max_shared[threadIdx.x] = order;
    }
    __syncthreads();

    for (unsigned int s = BlockSize/2; s >= 32; s>>=1)
    {
        if (threadIdx.x < s && max_shared[threadIdx.x+s] > max_shared[threadIdx.x])
            max_shared[threadIdx.x] = max_shared[threadIdx.x+s];
        __syncthreads();
    }

    GraphElem max = max_shared[threadIdx.x%32];
    for (int offset = 16; offset > 0; offset /= 2)
    {
        GraphElem tmp = __shfl_down_sync(0xffffffff, max, offset);
        max = (tmp > max) ? tmp : max;
    }

    if(threadIdx.x == 0)
        orders[blockIdx.x] = max; 
}

template<const int BlockSize>
__global__
void max_order_kernel
(
    GraphElem* __restrict__ orders, 
    GraphElem nv
)
{
    __shared__ GraphElem max_shared[BlockSize];

    max_shared[threadIdx.x] = 0;

    GraphElem u0 = threadIdx.x;

    for(GraphElem u = u0; u < nv; u += BlockSize)
    {
        GraphElem order = orders[u];
        if(max_shared[threadIdx.x] < order) 
            max_shared[threadIdx.x] = order;
    }
    __syncthreads();

    for (unsigned int s = BlockSize/2; s >= 32; s>>=1)
    {
        if (threadIdx.x < s && max_shared[threadIdx.x+s] > max_shared[threadIdx.x])
            max_shared[threadIdx.x] = max_shared[threadIdx.x+s];
        __syncthreads();
    }

    GraphElem max = max_shared[threadIdx.x%32];
    for (int offset = 16; offset > 0; offset /= 2)
    {
        GraphElem tmp = __shfl_down_sync(0xffffffff, max, offset);
        max = (tmp > max) ? tmp : max;
    }

    if(threadIdx.x == 0)
        orders[0] = max;

}

GraphElem max_order_cuda
(
    GraphElem* indices,
    GraphElem nv, 
    hipStream_t stream = 0  
)
{
    GraphElem* max_reduced;
    long long nblocks = (nv > MAX_GRIDDIM) ? MAX_GRIDDIM : nv;
    CudaMalloc(max_reduced, sizeof(GraphElem)*nblocks);
    max_order_reduce_kernel<BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(max_reduced, indices, nv);
    max_order_kernel<BLOCKDIM01><<<1,BLOCKDIM01, 0, stream>>>(max_reduced, nblocks);
    GraphElem max;
    CudaMemcpyAsyncDtoH(&max, max_reduced, sizeof(GraphElem), 0);
    CudaFree(max_reduced);
    return max;
}

template<const int BlockSize>
__global__
void move_index_orders_kernel
(
    GraphElem* __restrict__ dest, 
    GraphElem* __restrict__ src, 
    const GraphElem n
)
{
    const int i0 = threadIdx.x + BlockSize*blockIdx.x;
    for(GraphElem i = i0; i < n; i += BlockSize*gridDim.x)
        dest[i] = src[i]; 
}

void move_index_orders_cuda
(
    GraphElem* dest, 
    GraphElem* src, 
    const GraphElem& v0, 
    const GraphElem& v1, 
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem ne = e1-e0;
    long long nblocks = (ne > MAX_GRIDDIM) ? MAX_GRIDDIM : ne;
    move_index_orders_kernel<BLOCKDIM02><<<nblocks, BLOCKDIM02, 0, stream>>>(dest, src, ne);
}

template<const int WarpSize, const int BlockSize>
__global__
void reorder_edges_by_keys_kernel
(
    GraphElem* __restrict__ edges,
    GraphElem* indexOrders,
    GraphElem* __restrict__ indices,
    GraphElem* buff,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem ranges[(BlockSize/WarpSize)*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid  = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += (BlockSize/WarpSize)*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();

        GraphElem start = t_ranges[0]-e_base; 
        GraphElem end = t_ranges[1]-e_base;
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
        {
            GraphElem pos = indexOrders[i];
            buff[i] = edges[pos+start];
        }
        warp.sync();
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
            edges[i] = buff[i];
        warp.sync();
    } 
}

void reorder_edges_by_keys_cuda
(
    GraphElem* edges, 
    GraphElem* indexOrders, 
    GraphElem* indices, 
    GraphElem* buff, 
    const GraphElem& v0, 
    const GraphElem& v1,  
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    long long nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);
    reorder_edges_by_keys_kernel<WARPSIZE,BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>(edges, indexOrders, indices, buff, v0, e0, nv);
}

template<const int WarpSize, const int BlockSize>
__global__
void reorder_weights_by_keys_kernel
(
    GraphWeight* __restrict__ edgeWeights,
    GraphElem*   indexOrders,
    GraphElem*   __restrict__ indices,
    GraphWeight* buff,
    const GraphElem v_base,
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__ GraphElem ranges[(BlockSize/WarpSize)*2];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid  = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*2];

    GraphElem u0 = block_tid/WarpSize+BlockSize/WarpSize*blockIdx.x;
    for(GraphElem u = u0; u < nv; u += (BlockSize/WarpSize)*gridDim.x)
    {
        if(warp_tid < 2)
            t_ranges[warp_tid] = indices[u+warp_tid+v_base];
        warp.sync();

        GraphElem start = t_ranges[0]-e_base; 
        GraphElem end   = t_ranges[1]-e_base;
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
        {
            GraphElem pos = indexOrders[i];
            buff[i] = edgeWeights[pos+start];
        }
        warp.sync();
        for(GraphElem i = start+warp_tid; i < end; i += WarpSize)
            edgeWeights[i] = buff[i];
        warp.sync();
    } 
}

void reorder_weights_by_keys_cuda
( 
    GraphWeight* edgeWeights, 
    GraphElem* indexOrders, 
    GraphElem* indices , 
    GraphWeight* buff, 
    const GraphElem& v0, 
    const GraphElem& v1,  
    const GraphElem& e0, 
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    GraphElem nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);
    reorder_weights_by_keys_kernel<WARPSIZE,BLOCKDIM01><<<nblocks, BLOCKDIM01,0,stream>>>(edgeWeights, indexOrders, indices, buff, v0, e0, nv);
}
#if 0
template<const int WarpSize, const int BlockSize>
__global__
void build_local_commid_offset_kernel
(
    GraphElem* offset,
    GraphElem* edges,
    GraphElem* indices,
    GraphElem* commIds,
    GraphElem nv
)
{
    
}

void build_local_commid_offset_cuda
(
)
{

}

void compute_vertex_self_weight
(
    GraphWeight* vertexSelfWeights,
    GraphElem*   edges,
    GraphWeight* edgesWeights,
    GraphElem* indices,
    const GraphElem& nv,
    cosnt GraphElem& ne,
    hipStream_t stream =0
)
{

}
#endif
#if 0
void compute_modularity_cuda
(
    GraphWeight* vertexSelfWeights,
    GraphWeight* commWeights,
    GraphWeight* m,
    const GraphElem& nv
)
{

}
#endif
#if 0
//#else

template<const int WarpSize, const int BlockSize>
__global__
void fill_edges_community_ids_kernel
(
    GraphElem2* __restrict__ commIdKeys,
    GraphElem*  __restrict__ edges,
    GraphElem*  __restrict__ indices,
    GraphElem*  __restrict__ commIds,
    const GraphElem v_base, 
    const GraphElem e_base,
    const GraphElem nv
)
{
    __shared__  GraphElem ranges[BlockSize];

    cg::thread_block block = cg::this_thread_block();
    cg::thread_group warp = cg::tiled_partition<WarpSize>(block);

    const unsigned block_tid = block.thread_rank();
    const unsigned warp_tid = warp.thread_rank();

    GraphElem* t_ranges = &ranges[(block_tid/WarpSize)*WarpSize];

    GraphElem nu = (nv + gridDim.x-1) / gridDim.x;
    GraphElem u0 = blockIdx.x*nu;
    GraphElem u1 = u0 + nu;
    if(u1 > nv) 
        u1 = nv;
    GraphElem start, end;
    start = indices[u0+v_base]-e_base;
    for(GraphElem u = 0; u < nu; ++u)
    {
        if(u%WarpSize == 0)
        {
            warp.sync();
            if(u+warp_tid <= nu)
                t_ranges[warp_tid] = indices[u0+u+warp_tid+v_base+1];
            warp.sync();
        }
        end = t_ragnes[u%WarpSize]-e_base;
        for(GraphElem i = start+warp_id; i < end; i += WarpSize)
        {
            GraphElem commId = commIds[edges[i]];    
            #ifdef USE_32BIT_GRAPH
            commIdKeys[i] = make_int2(u, commId);
            #else
            commIdKeys[i] = make_longlong2(u, commId);
            #endif
        }
        start = end;
    } 
}

void fill_edges_community_ids_cuda
(
    GraphElem2* commIdKeys, 
    GraphElem* edges,
    GraphElem* indices,
    GraphElem* commIds,
    const GraphElem& v0,
    const GraphElem& v1,
    const GraphElem& e0,
    const GraphElem& e1,
    hipStream_t stream = 0
)
{
    GraphElem nv = v1-v0;
    long long nblocks = (nv/(BLOCKDIM01/WARPSIZE) > MAX_GRIDDIM) ? MAX_GRIDDIM : nv/(BLOCKDIM01/WARPSIZE);    
    fill_edges_community_ids_kernel<WARPSIZE, BLOCKDIM01><<<nblocks, BLOCKDIM01, 0, stream>>>
    (commIdKeys, edges, indices, commIds, v0, e0, nv);
}
#endif
